#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <unistd.h>
#include <time.h>

#include "tiffio.h"

#include "integrator.h"
#include "gl_helper.h"

//#define DEBUG

// global constants
/* watch saves the step number to be controlled 
 */
unsigned w, h, n_loop, loop_done, watch;
float heating_level = 0;
dim3 block_num, thread_num;
size_t size_shared;
float *dT;
float *T_device, *K_device, *dT_device, *tmp;
int *operation, *d_operation;	// Serve a controllare che il programma ricalcoli tutti i punti
uchar4 *image;
double cpu_time, cpu_step;
size_t temp_size, op_size;


void readTiff(char *filename, float **raster, unsigned *w, unsigned *h, 
	      float scale)
{
	// Open file
	TIFF* img = TIFFOpen(filename, "r");
	
	// read file size
	TIFFGetField(img, TIFFTAG_IMAGEWIDTH, w);
	TIFFGetField(img, TIFFTAG_IMAGELENGTH, h);

	// allocate memory
	uint32 *tmp_raster = (uint32*) _TIFFmalloc((*w)*(*h) * sizeof(uint32));
	*raster = (float *) malloc((*w)*(*h) * sizeof(float));

	if (tmp_raster != NULL) {
		// finally read image
	    if (TIFFReadRGBAImage(img, *w, *h, tmp_raster, 0)) {
			// separate red channnel (we have grayscale images so it's the same)
			// and convert to float with scale
			unsigned i;
			for (i = 0; i < (*w)*(*h); ++i) {
				(*raster)[i] = ((float)TIFFGetR(tmp_raster[i])) * scale;
			}
	    }
	    _TIFFfree(tmp_raster);
	} else {
		printf("ERROR: cannot read file '%s'", filename);
	}

	TIFFClose(img);
}

void interpolate_array(float *in, float *out, unsigned size, float opacity)
{
    unsigned i;
    for (i = 0; i < size; ++i) {
        out[i] = in[i] * opacity;
	}
}

void on_key_pressed(unsigned char key, int x, int y)
{
  /* a cosa servono x e y? 
   * gli if sono giusti? nel secondo if cambio heating_level>0 in >1 
   * detto ciö credo sia meglio impostare l´heating_level a 1.1 se ´+´, 0.9 se ´-´
   * inoltre credo che occorre passare tmp al posto di dT, perché tmp é passato come out
   * dT come in alla funyione interpolate_array
   * di sotto un esempio:
     switch(key) {
        case '+':
            heating_level = 1.1
            unsigned size = w * h;
            interpolate_array(dT, tmp, size, heating_level);
            hipMemcpy(dT_device, tmp, size * sizeof(float), hipMemcpyHostToDevice);
            break;
        case '-':
            unsigned size = w * h;
            heating_level 0.9;
            interpolate_array(dT, tmp, size, heating_level);
            hipMemcpy(dT_device, tmp, size * sizeof(float), hipMemcpyHostToDevice);
            break;
    }*/
  
  	unsigned size = w * h;
    switch(key) {
        case '+':
            if (heating_level < 0.95) {
                heating_level += 0.1;
                interpolate_array(dT, tmp, size, heating_level);
                hipMemcpy(dT_device, tmp, size * sizeof(float), hipMemcpyHostToDevice);
            }
            break;
        case '-':
            if (heating_level > 0.05) {
                heating_level -= 0.1;
                interpolate_array(dT, tmp, size, heating_level);
                hipMemcpy(dT_device, tmp, size * sizeof(float), hipMemcpyHostToDevice);
            }
            break;
    }

	char title[257];
	sprintf(title, "Heat equation (heating: %.0f %% - Avg (ms): %.1f)",
		heating_level * 100, cpu_time / loop_done * 1000);

	glutSetWindowTitle(title);
}

void step()
{
	//printf("dT: %p\n", dT);
	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

	hipArray *tex = map_texture();
	clock_t start_host, end_host; // Used to check time of execution
	
	// Copy data for controlling the correct execution of the simulation
	/*float *T_check;
	int i;
	T_check = (float*)malloc(temp_size);
	
	if (loop_done == watch){
	  	hipMemcpy(T_check, T_device, temp_size, hipMemcpyDeviceToHost);	

	  	FILE *ftemp ;
	  	ftemp = fopen("check/temperature.txt", "w");
	  	if (ftemp == NULL){
	    	printf("\nError while opening file temperature.txt\n");
	    	perror("Error while opnening file temperature.txt");
	    	exit(1);
	  	}
	  
	  	fprintf(ftemp, "row\n");
	  	for (i=514*257; i<514*257+514; i++){
	      	fprintf(ftemp, "%f\n", T_check[i]);
	  	}
	  	fprintf(ftemp, "\n\n\ncolumn:\n");
	  	for (i=0; i<514; i++){
	    	  fprintf(ftemp, "%f\n", T_check[257+i*514]);
	  	}
	  	fclose(ftemp);

	  	int j;
	  	FILE *fgrid;
	  	fgrid = fopen("check/T_step.txt", "w");
	  	if (fgrid == NULL){
	    	printf("\nError while opening file T_step.txt\n");
	    	perror("Error while opnening file T_step.txt");
	    	exit(1);
	  	}
	  	for (i=0; i<514; i++){
	  		for (j=0; j<514; j++){
	      		fprintf(fgrid, "%f ", T_check[i*514 + j]);
	    	}
			if (i != 513)
	      		fprintf(fgrid, "\n ");
	  	}
	  	fclose(fgrid);

	  	hipMemcpy(operation, d_operation, temp_size, hipMemcpyDeviceToHost);
	  	FILE *fop;
	  	fop = fopen("check/operation.txt", "w");
	  	if (fop == NULL){
	    	printf("\nError while opening file T_step.txt\n");
	    	perror("Error while opnening file T_step.txt");
	    	exit(1);
	  	}
	  	for (i=0; i<514; i++){
	    	for (j=0; j<514; j++){
	      		fprintf(fop, "%d ", operation[i*514 + j]);
	    	}
			if (i != 513)
	      		fprintf(fgrid, "\n ");
	  	}
	  	fclose(fop);
	}*/

	// START SIMULATION
	start_host = clock();
	unsigned z;
	for (z = 0; z < 24; ++z) {
		stepSimulation2D<<<block_num, thread_num, size_shared>>>
			(T_device, K_device, dT_device, n_loop, image, d_operation);
	}
	hipError_t error = hipDeviceSynchronize();
	end_host=clock();

	if (error != hipSuccess) {
		printf("Error while running kernel: %s\n", hipGetErrorString(error));
	}
	
	cpu_step = ((double)  (end_host - start_host));
	cpu_time += cpu_step / CLOCKS_PER_SEC;
	++loop_done;

	// Print time statistics
//#ifdef DEBUG	
	FILE *ftime;
	if (loop_done == watch){
		ftime = fopen("check/mean_time.txt", "a");
		if (ftime == NULL){
		  	printf("\nError while opening file mean_time.txt\n");
		  	perror("Error while opnening file mean_time.txt");
		  	exit(1);
		}
		fprintf(ftime, "\nNumber of blocks: %d\n", block_num.x);
		fprintf(ftime, "Total Time: %f\nMean Time per Step: %f\n", cpu_time, 
				cpu_time/(double)loop_done);
		fclose(ftime);
		
		printf("Time saved\n");
	}
	ftime = fopen("check/exe_time.txt", "a");
	if (ftime == NULL){
	  	printf("\nError while opening file mean_time.txt\n");
	  	perror("Error while opnening file mean_time.txt");
	  	exit(1);
	}
	fprintf(ftime, "%f\n", cpu_step / CLOCKS_PER_SEC);
	fclose(ftime);
//#endif

	// This copies image to texture
	hipMemcpyToArray(tex, 0, 0, image, w*h*4, hipMemcpyDeviceToDevice);

	unmap_and_draw();

	glutSwapBuffers();
	glutPostRedisplay();
}

int main(int argc, char **argv)
{
	// First check if a directory was given
	if (argc < 2) {
	    printf("No simulation given.\n");
		return -1;
	}

	// Build filenames
	char *simulation_folder = argv[1];
	unsigned len = strlen(simulation_folder);
	char temperature[257];
	char conductivity[257];
	char heating[257];

	strcpy(temperature, simulation_folder);
	strcpy(temperature + len, "temperature.tiff");
	strcpy(conductivity, simulation_folder);
	strcpy(conductivity + len, "conductivity.tiff");
	strcpy(heating, simulation_folder);
	strcpy(heating + len, "heating.tiff");

	// read files
	float *T, *K, dt;
	dt = 0.0001;
	readTiff(temperature, &T, &w, &h, 1);
	readTiff(conductivity, &K, &w, &h, dt);	
	// 0.01 is unstable, 0.001 is the first stable 
	readTiff(heating, &dT, &w, &h, dt);
	// if scale factor too high temperature overflow
	printf("Simulation size: %ux%u\n", w, h);
	
	// Setup other interesting stuff and parse other command line arguments
	unsigned block_side = 8;
	n_loop = 8;
	if (argc > 2) {
	    unsigned j;
	    for (j = 2; j < argc; ++j) {
	        if (!strcmp(argv[j], "-bn")) {
		        sscanf(argv[j+1], "%u", &block_side);
	        } else if (!strcmp(argv[j], "-n")) {
		        sscanf(argv[j+1], "%u", &n_loop);
	        } else if (!strcmp(argv[j], "-l")) {
		        sscanf(argv[j+1], "%u", &watch);
	        }
	    }
	}
	printf("-------------------------\n");
	printf("Loops per thread: %u\n", n_loop);
	printf("Block size: %ux%u (%u threads per block)\n",
	    w/(block_side), w/(block_side*n_loop), w*w / (block_side*block_side*n_loop));
	
	// for heating
	size_t param_size = w * h * sizeof(float);
	temp_size = (w + 2) * (h + 2) * sizeof(float);
	op_size = (w + 2) * (h + 2) * sizeof(int);
	tmp = (float *) malloc(param_size);
	interpolate_array(dT, tmp, w*h, heating_level);
	
	// dimensions of grid, blocks and shared memory
	thread_num.x = w/(block_side*n_loop);
	thread_num.y = w/(block_side);
	block_num.x = block_side;
	block_num.y = block_side;
	size_shared = sizeof(float) * (w/(block_side) + 2) * (w/(block_side) + 2);
	
	printf("Grid size: %ux%u\n", block_num.x, block_num.y);
	printf("Shared memory: %.2f Kb\n", size_shared / 1024.f);
	
	// Copy input to device 
	hipMalloc(&T_device, temp_size);
	hipMemcpy(T_device, T, temp_size, hipMemcpyHostToDevice);
	
	hipMalloc(&K_device, param_size);
	hipMemcpy(K_device, K, param_size, hipMemcpyHostToDevice);
	
	hipMalloc(&dT_device, param_size);
	hipMemcpy(dT_device, tmp, param_size, hipMemcpyHostToDevice);

	hipMalloc(&d_operation, op_size);
	hipMemcpy(d_operation, operation, op_size, hipMemcpyHostToDevice);

	hipMalloc(&image, w*h*4);

	// Now that we are done loading the simulation, we start OpenGL
	initGL(&argc, argv, "Heat equation", step, 512, 512);

	hipSetDevice(0);
	cudaGLSetGLDevice(0);

	register_texture(w, h);
	glutKeyboardFunc(on_key_pressed);
	
	//register_array(n * 2, sizeof(float), n);

	// Start simulation
	loop_done = 0;
	cpu_time = 0;
	glutMainLoop();
	
	// Looks like code after glutMainLoop(); doesn´t work... 	

	// cleanup
	free(T);
	free(K);
	free(dT);
	free(tmp);
	hipFree(T_device);
        
    return 0;
}
