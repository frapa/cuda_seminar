#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <unistd.h>
#include <time.h>

#include "tiffio.h"

#include "integrator.h"
#include "gl_helper.h"

#define DEBUG

// global constants
/* watch saves the step number to be controlled 
 */
unsigned w, h, n_loop, loop_done, watch;
float heating_level = 0;
dim3 block_num, thread_num;
size_t size_shared;
float *dT;
float *T_device, *K_device, *dT_device, *tmp;
uchar4 *image;
double cpu_time, cpu_step;
size_t temp_size;

void readTiff(char *filename, float **raster, unsigned *w, unsigned *h, float scale)
{
	// Open file
	TIFF* img = TIFFOpen(filename, "r");
	
	// read file size
	TIFFGetField(img, TIFFTAG_IMAGEWIDTH, w);
	TIFFGetField(img, TIFFTAG_IMAGELENGTH, h);

	// allocate memory
	uint32 *tmp_raster = (uint32*) _TIFFmalloc((*w)*(*h) * sizeof(uint32));
	*raster = (float *) malloc((*w)*(*h) * sizeof(float));

	if (tmp_raster != NULL) {
		// finally read image
	    if (TIFFReadRGBAImage(img, *w, *h, tmp_raster, 0)) {
			// separate red channnel (we have grayscale images so it's the same)
			// and convert to float with scale
			unsigned i;
			for (i = 0; i < (*w)*(*h); ++i) {
				(*raster)[i] = ((float)TIFFGetR(tmp_raster[i])) * scale;
			}
	    }
	    _TIFFfree(tmp_raster);
	} else {
		printf("ERROR: cannot read file '%s'", filename);
	}

	TIFFClose(img);
}

void interpolate_array(float *in, float *out, unsigned size, float opacity)
{
    unsigned i;
    for (i = 0; i < size; ++i)
        out[i] = in[i] * opacity;
}

void on_key_pressed(unsigned char key, int x, int y)
{
  /* a cosa servono x e y? 
   * gli if sono giusti? nel secondo if cambio heating_level>0 in >1 
   * detto ciö credo sia meglio impostare l´heating_level a 1.1 se ´+´, 0.9 se ´-´
   * inoltre credo che occorre passare tmp al posto di dT, perché tmp é passato come out
   * dT come in alla funyione interpolate_array
   * di sotto un esempio:
     switch(key) {
        case '+':
            heating_level = 1.1
            unsigned size = w * h;
            interpolate_array(dT, tmp, size, heating_level);
            hipMemcpy(dT_device, tmp, size * sizeof(float), hipMemcpyHostToDevice);
            break;
        case '-':
            unsigned size = w * h;
            heating_level 0.9;
            interpolate_array(dT, tmp, size, heating_level);
            hipMemcpy(dT_device, tmp, size * sizeof(float), hipMemcpyHostToDevice);
            break;
    }*/
  
    switch(key) {
        case '+':
            if (heating_level < 1) {
                unsigned size = w * h;
                heating_level += 0.1;
                interpolate_array(dT, tmp, size, heating_level);
                hipMemcpy(dT_device, dT, size * sizeof(float), hipMemcpyHostToDevice);
            }
            break;
        case '-':
            if (heating_level > 1) {
                unsigned size = w * h;
                heating_level -= 0.1;
                interpolate_array(dT, tmp, size, heating_level);
                hipMemcpy(dT_device, dT, size * sizeof(float), hipMemcpyHostToDevice);
            }
            break;
    }
}

void step()
{
	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

	hipArray *tex = map_texture();
	clock_t start_host, end_host; // Used to check time of execution
	
	// Copy data for controlling the correct execution of the simulation
	float *T_check;
	int i;
	T_check = (float*)malloc(temp_size);
	
	if (loop_done == watch){
	  hipMemcpy(T_check, T_device, temp_size, hipMemcpyDeviceToHost);	
	  
	  FILE *ftemp ;
	  ftemp = fopen("check/temperature.txt", "w");
	  if (ftemp == NULL){
	    printf("\nError while opening file temperature.txt\n");
	    perror("Error while opnening file temperature.txt");
	    exit(1);
	  }
	  
	  for (i=514*257; i<514*257+514; i++){
	      fprintf(ftemp, "%f\n", T_check[i]);
	  }
	  fprintf(ftemp, "\n\n\n");
	  for (i=0; i<514; i++){
	      fprintf(ftemp, "%f\n", T_check[257+i*514]);
	  }
	  /*
	  int j;
	  FILE *fgrid;
	  fgrid = fopen("check/T_step.txt", "w");
	  if (ftemp == NULL){
	    printf("\nError while opening file T_step.txt\n");
	    perror("Error while opnening file T_step.txt");
	    exit(1);
	  }
	  for (i=0; i<h; i++){
	    for (j=0; j<w; j++){
	      fprintf(fgrid, "%f ", T_check[i*w + j]);
	    }
	      fprintf(fgrid, "\n ");
	  }
	  fclose(fgrid);*/
	  fclose(ftemp);
	}

	
	start_host = clock();
	stepSimulation2D<<<block_num, thread_num, size_shared>>>
	    (T_device, K_device, dT_device, n_loop, image);
	hipError_t error = hipDeviceSynchronize();
	end_host=clock();

	if (error != hipSuccess) {
		printf("Error while running kernel: %s\n", hipGetErrorString(error));
	}
	
	cpu_step = ((double)  (end_host - start_host));
	cpu_time += cpu_step / CLOCKS_PER_SEC;
	loop_done += 1;

	hipMemcpyToArray(tex, 0, 0, image, w*h*4, hipMemcpyDeviceToDevice);

	unmap_and_draw();
	
	glutSwapBuffers();
	glutPostRedisplay();
}

int main(int argc, char **argv)
{
	// First check if a directory was given
	if (argc < 2) {
	    printf("No simulation given.\n");
		return -1;
	}

	// Build filenames
	char *simulation_folder = argv[1];
	unsigned len = strlen(simulation_folder);
	char *temperature = (char *)malloc(len + 17);
	char *conductivity = (char *)malloc(len + 18);
	char *heating = (char *)malloc(len + 13);

	strcpy(temperature, simulation_folder);
	strcpy(temperature + len, "temperature.tiff");
	strcpy(conductivity, simulation_folder);
	strcpy(conductivity + len, "conductivity.tiff");
	strcpy(heating, simulation_folder);
	strcpy(heating + len, "heating.tiff");

	// read files
	float *T, *K, *dT;
	readTiff(temperature, &T, &w, &h, 1);
	readTiff(conductivity, &K, &w, &h, 0.1);	// Previous: 0.0001
	readTiff(heating, &dT, &w, &h, 1);
	printf("Simulation size: %ux%u\n", w, h);
	
	// check input
	int i;
	FILE *ftemp ;
	ftemp = fopen("check/initial.txt", "w");
	if (ftemp == NULL){
	  printf("\nError while opening file initial.txt\n");
	  perror("Error while opnening file initial.txt");
	  exit(1);
	}
	for (i=514*257; i<514*257+514; i++){
	    fprintf(ftemp, "%f\n", T[i]);
	}
	fprintf(ftemp, "\n\n\n");
	for (i=0; i<514; i++){
	    fprintf(ftemp, "%f\n", T[257+i*514]);
	}
	/*
	int j
	FILE *fgrid;
	fgrid = fopen("check/T_initial.txt", "w");
	if (ftemp == NULL){
	  printf("\nError while opening file T_initial.txt\n");
	  perror("Error while opnening file T_initial.txt");
	  exit(1);
	}
	for (i=0; i<h; i++){
	  for (j=0; j<w; j++){
	    fprintf(fgrid, "%f ", T[i*w + j]);
	  }
 	  fprintf(fgrid, "\n");
	}
	fclose(fgrid);*/
	fclose(ftemp);
	
	
	// Setup other interesting stuff
	// Parse other command line arguments
	unsigned square_side = 64;
	n_loop = 8;
	if (argc > 2) {
	    unsigned j;
	    for (j = 2; j < argc; ++j) {
	        if (!strcmp(argv[j], "-bs")) {
		        sscanf(argv[j+1], "%u", &square_side);
	        } else if (!strcmp(argv[j], "-n")) {
		        sscanf(argv[j+1], "%u", &n_loop);
	        } else if (!strcmp(argv[j], "-l")) {
		        sscanf(argv[j+1], "%u", &watch);
	        }
	    }
	}
	printf("-------------------------\n");
	printf("Loops per thread: %u\n", n_loop);
	printf("Block size: %ux%u (%u threads per block)\n",
	    square_side, square_side/n_loop, square_side*square_side/n_loop);
	
	// for heating
	size_t param_size = w * h * sizeof(float);
	temp_size = (w + 2) * (h + 2) * sizeof(float);
	tmp = (float *) malloc(param_size);
	
	// dimensions of grid, blocks and shared memory
	thread_num.x = square_side / n_loop;
	thread_num.y = square_side;
	block_num.x = w / square_side;	// non sarebbe piú semplice impostarlo ad n_loop?
	block_num.y = h / square_side;
	size_shared = sizeof(float) * (square_side + 2) * (square_side + 2);
	
	printf("Grid size: %ux%u\n", block_num.x, block_num.y);
	// Kilobit (Kb)? non Kilobyte (KB)? 
	printf("Shared memory: %.2f Kb\n", size_shared / 1024.f);
	
	// Copy input to device 
	hipMalloc(&T_device, temp_size);
	hipMemcpy(T_device, T, temp_size, hipMemcpyHostToDevice);
	
	hipMalloc(&K_device, param_size);
	hipMemcpy(K_device, K, param_size, hipMemcpyHostToDevice);
	
	hipMalloc(&dT_device, param_size);
	hipMemcpy(dT_device, dT, param_size, hipMemcpyHostToDevice);

	hipMalloc(&image, w*h*4);

	// Now that we are done loading the simulation, we start OpenGL
	initGL(&argc, argv, "Heat equation", step);

	hipSetDevice(0);
	cudaGLSetGLDevice(0);

	register_texture(w, h);
	glutKeyboardFunc(on_key_pressed);
	
	//register_array(n * 2, sizeof(float), n);

	// Start simulation
	loop_done = 0;
	cpu_time = 0;
	glutMainLoop();
	
	// Print time statistics DOES´N WORK! FIGURE OUT WHY
	printf("Total Time: %f\n", cpu_time);
	printf("Mean Time per Step: %f\n", cpu_time/(double)loop_done);
	fflush(stdout);
	
	// cleanup
	free(T);
	free(K);
	free(dT);
	free(tmp);
	hipFree(T_device);
        
    return 0;
}
